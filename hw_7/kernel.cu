/**
* University of Pittsburgh
* Department of Computer Science
* CS1645: Introduction to HPC Systems
* Instructor: Xiaolong Cui
* This is a skeleton for implementing prefix sum using GPU, inspired
* by nvidia course of similar name.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define N 512

/* Prototypes */
void random_floats(float* a, int n);
void serial_scan(float* out, float* in, int n);

/*
* You should implement the parallel scan function here!
*/
__global__ void parallel_scan(float *g_odata, float *g_idata, int n) 
{
	extern __shared__ float temp_buf[];  
	int thread_name = threadIdx.x;
	int out_place = 0;
	int	in_place = 1;
	
	temp_buf[out_place*n + thread_name] = (thread_name > 0) ? g_idata[thread_name - 1] : 0;
	/* Make sure threads are synchronized on transfer*/
	__syncthreads();
	for (int displacement = 1; displacement < n; displacement = displacement << 1) {
		
		out_place = 1 - out_place; 
		in_place = 1 - out_place;
		
		if (thread_name >= displacement) {
			temp_buf[out_place*n + thread_name] += temp_buf[in_place*n + thread_name - displacement];
		}
		else {
			temp_buf[out_place*n + thread_name] = temp_buf[in_place*n + thread_name];
		}
		__syncthreads();
	}
	g_odata[thread_name] = temp_buf[out_place*n + thread_name]; 
}



/*
* Fills an array a with n random floats.
*/
void random_floats(float* a, int n) {
	float d;
	// Comment out this line if you want consistent "random".
	srand(time(NULL));
	for (int i = 0; i < n; ++i) {
		d = rand() % 8;
		a[i] = ((rand() % 64) / (d > 0 ? d : 1));
	}
}

/*
* Simple Serial implementation of exclusive scan.
*/
void serial_scan(float* out, float* in, int n) {
	float total_sum = 0;
	out[0] = 0;
	for (int i = 1; i < n; i++) {
		total_sum += in[i - 1];
		out[i] = out[i - 1] + in[i - 1];
	}
	if (total_sum != out[n - 1]) {
		printf("Warning: exceeding accuracy of float.\n");
	}
}

/*
* This is a simple function that confirms that the output of the scan
* function matches that of a golden image (array).
*/
bool printError(float *gold_out, float *test_out, bool show_all) {
	bool firstFail = true;
	bool error = false;
	float epislon = 0.1;
	float diff = 0.0;
	for (int i = 0; i < N; ++i) {
		diff = abs(gold_out[i] - test_out[i]);
		if ((diff > epislon) && firstFail) {
			printf("ERROR: gold_out[%d] = %f != test_out[%d] = %f // diff = %f \n", i, gold_out[i], i, test_out[i], diff);
			firstFail = show_all;
			error = true;
		}
	}
	return error;
}

int main(void) {
	float *in, *out, *gold_out; // host
	float *d_in, *d_out, *dgold_out; // device

	int size = sizeof(float) * N;

	in = (float *)malloc(size);
	random_floats(in, N);
	out = (float *)malloc(size);
	gold_out = (float *)malloc(size);

	/* Allocate arrays on the device*/
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);
	hipMalloc((void **)&dgold_out, size);

	/*Copy over the input array to the device*/
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

	// ***********
	// RUN SERIAL SCAN
	// ***********
	serial_scan(gold_out, in, N);

	// ***********

	// RUN PARALLEL SCAN
	// ***********
	parallel_scan<<<1,N>>>(dgold_out, d_in, N);


	if (printError(gold_out, out, false)) {
		printf("ERROR: The parallel scan function failed to produce proper output.\n");
	}
	else {
		printf("CONGRATS: The parallel scan function produced proper output.\n");
	}



	return 0;
}
